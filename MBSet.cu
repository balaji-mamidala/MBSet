#include "hip/hip_runtime.h"
/* 
 * Created on June 24, 2012
 *
 * Purpose:  This program displays Mandelbrot set using the GPU via CUDA and
 * OpenGL immediate mode.
 *
 */

#include <iostream>
#include <stack>
#include <hip/hip_runtime_api.h>
#include <stdio.h>
#include "Complex.cu"

#include <GL/freeglut.h>

#include <vector>
//#include <math.h>

#define THREADS_PER_BLOCK     32

using namespace std;

const uint16_t maxIt = 2000; // Maximum Iterations

// Size of window in pixels, both width and height
uint16_t WINDOW_DIM_X = 512;
uint16_t WINDOW_DIM_Y = 512;

uint16_t SIZE_X = WINDOW_DIM_X;
uint16_t SIZE_Y = WINDOW_DIM_Y;
uint32_t SIZE = SIZE_X*SIZE_Y;

// Initial screen coordinates, both host and device.
Complex minC(-2.0, -1.2);
Complex maxC(1.0, 1.8);

vector<Complex*> vec_c; //Vector that stores zoomed c values
Complex *c;
Complex *d_c;
Complex *d_minC;
Complex *d_maxC;


vector<int*> vec_MBSet; //Vector that stores zoomed sets
int *MBSet; //MBSet for current frame
int *d_MBSet;

uint16_t x1draw, y1draw, x2draw, y2draw;

bool draw_box = false;
bool left_click = false;

enum color_base {red, red_orange, metallic_gold, gold};
color_base use_color = red;

int windowID;


// Define the RGB Class
class RGB
{
public:
  RGB()
    : r(0), g(0), b(0) {}
  RGB(double r0, double g0, double b0)
    : r(r0), g(g0), b(b0) {}
public:
  double r;
  double g;
  double b;
};


RGB* colors = new RGB[maxIt + 1];

void InitializeColors()
{
  //Use histogram approach for color
  uint16_t histogram[maxIt+1];

  for(uint16_t i=0; i<maxIt+1; i++)
  {
    histogram[i] = 0;
  }

  for(uint32_t i=0; i<SIZE; i++)
  {
    histogram[MBSet[i]]++;
  }

  uint64_t total_sum = 0;
  for(uint16_t i=0; i<maxIt+1; i++)
  {
    total_sum += histogram[i];
  }


  float sred = 0.0;
  float ered;
  float sgreen = 0.0;
  float egreen;
  float sblue = 0.0;
  float eblue;

  //Red
  if(red == use_color)
  {
    ered = 0.9;
    egreen = 0.0;
    eblue = 0.0;
  }
  else if(red_orange == use_color)
  {
    ered = 1.0;
    egreen = 0.4;
    eblue = 0.0;
  }
  else if(metallic_gold == use_color)
  {
    ered = 0.828;
    egreen = 0.6853;
    eblue = 0.214;
  }
  else if(gold == use_color)
  {
    ered = 1.0;
    egreen = 0.84;
    eblue = 0.0;
  }

  uint64_t running_total = 0;
  for (int i = 0; i < maxIt; ++i)
  {
    running_total += histogram[i];

    float percent = running_total/(float)total_sum;

    float r = sred + ((ered - sred) * percent);
    float g = sgreen + ((egreen - sgreen) * percent);
    float b = sblue + ((eblue - sblue) * percent);

    colors[i] = RGB(r, g, b);
  }

  colors[maxIt] = RGB(); // black
}


/*void window_data_init()
{
  for(uint16_t row=0; row < WINDOW_DIM_X; row++)
  {
    for(uint16_t col=0; col< WINDOW_DIM_Y; col++)
    {
      c[(row*SIZE_X) + col].r =  minC.r + (rstep * row);
      c[(row*SIZE_X) + col].i =  minC.i + (istep * col);
    }
  }
}*/


__global__ void calcMBS_limit(Complex* minC, Complex* maxC, Complex* c, int* MBSet)
{
  uint32_t index = threadIdx.x + (blockIdx.x * blockDim.x);

  int row = index/512;
  int col = index - (row*512);

  float rstep = (maxC[0].r - minC[0].r)/(float)512; 
  float istep = (maxC[0].i - minC[0].i)/(float)512;

  c[index].r = minC[0].r + (rstep * row);
  c[index].i = minC[0].i + (istep * col);

  Complex Z(c[index].r, c[index].i);

  for(MBSet[index]=0; MBSet[index] < maxIt ; MBSet[index]++)
  {
    Z = (Z*Z) + c[index];

    if(Z.magnitude2() > 4.0)
    {
      break;
    }
  }
}


///Opengl functions begin
void drawModel()
{ 
  //Display MBset
  glBegin(GL_POINTS);
  uint32_t index;
  for(uint16_t row=0; row<WINDOW_DIM_X; row++)
  {
    for(uint16_t col=0; col<WINDOW_DIM_Y; col++)
    {
      index = row * SIZE_X + col;
      glColor3d(colors[MBSet[index]].r, colors[MBSet[index]].g, colors[MBSet[index]].b);
      glVertex2i(row, WINDOW_DIM_Y - col -1 );
    }
  }
  glEnd();

  if(true == draw_box)
  {
    glColor3f(1.0, 0.0, 0.0);
    glScalef(1.0, 1.0, 0);
    glLineWidth(2.0);

    glBegin(GL_LINE_LOOP);

    glVertex2i(x1draw, WINDOW_DIM_Y - y1draw); // Top left corner
    glVertex2i(x2draw, WINDOW_DIM_Y - y1draw); // Top right corner
    glVertex2i(x2draw, WINDOW_DIM_Y - y2draw); // Bottom right corner
    glVertex2i(x1draw, WINDOW_DIM_Y - y2draw); // Bottom left corner

    glEnd();
  } 
}


void display(void)
{
  glClearColor(1.0, 1.0, 1.0, 1.0);
  glClear(GL_COLOR_BUFFER_BIT);
  glLoadIdentity();

  // Draw the model
  drawModel();
  // Swap the double buffers
  glutSwapBuffers();
}

void reshape(int w, int h)
{
  glViewport(0, 0, (GLsizei)WINDOW_DIM_X, (GLsizei)WINDOW_DIM_Y);
  glMatrixMode(GL_PROJECTION);
  glLoadIdentity();
  glOrtho(0.0, (GLdouble)WINDOW_DIM_X, (GLdouble)0.0, (GLdouble)WINDOW_DIM_X, ((GLdouble)-1), (GLdouble)1);
  glMatrixMode(GL_MODELVIEW);
  glLoadIdentity();

  glutReshapeWindow(WINDOW_DIM_X, WINDOW_DIM_Y);

  glutPostRedisplay();   // repaint the window
}


void keyboard(unsigned char key, int x, int y)
{
  switch(key)
  {
    case 'q':
      glutDestroyWindow(windowID);
      exit(0);

    case 'b':
      if( (0!= vec_c.size()) && (0!= vec_MBSet.size()) )
      {
        free(c);
        free(MBSet);

        c = vec_c.back();
        MBSet = vec_MBSet.back();

        vec_c.pop_back();
        vec_MBSet.pop_back();
      
        InitializeColors();
        glutPostRedisplay();  
      }

      break;
    
    case 'c':
      if(red == use_color)
      {
        use_color = red_orange;
      }
      else if(red_orange == use_color)
      {
        use_color = metallic_gold;
      }
      else if(metallic_gold == use_color)
      {
        use_color = gold;
      }
      else if(gold == use_color)
      {
        use_color = red;
      }

      InitializeColors();
      glutPostRedisplay();   // repaint the window
      break;   

    default:
      break;
  }
}

void order_co_ordinate(void)
{

  //cout << "Before re-order :: " << "X1: " << x1draw << " Y1: " << y1draw << " X2: " << x2draw << " Y2: " << y2draw << endl;

  if(x1draw > x2draw)
  {
    uint16_t temp = x1draw;
    x1draw = x2draw;
    x2draw = temp;
  }

  if(y1draw > y2draw)
  {
    uint16_t temp = y1draw;
    y1draw = y2draw;
    y2draw = temp;
  }

  //cout << "After re-order :: " << "X1: " << x1draw << " Y1: " << y1draw << " X2: " << x2draw << " Y2: " << y2draw << endl;
}


void re_calc_draw_MBSet(void)
{
  order_co_ordinate();

  uint32_t min_index = (x1draw * SIZE_X) + y1draw;
  uint32_t max_index = (x2draw * SIZE_X) + y2draw;

  vec_c.push_back(c);
  vec_MBSet.push_back(MBSet);

  minC.r = c[min_index].r;
  minC.i = c[min_index].i;

  maxC.r = c[max_index].r;
  maxC.i = c[max_index].i;

  //Allocate more memory for new arrays
  c = (Complex*) malloc(SIZE*sizeof(Complex));
  MBSet = (int*) malloc(SIZE*sizeof(int));

  //Copy from Host to device
  hipMemcpy(d_minC, &minC, sizeof(Complex), hipMemcpyHostToDevice);
  hipMemcpy(d_maxC, &maxC, sizeof(Complex), hipMemcpyHostToDevice);
  hipMemcpy(d_MBSet, MBSet, SIZE*sizeof(int), hipMemcpyHostToDevice);

  // Calculate the interation counts
  //Launch MBS calculation on GPU
  calcMBS_limit<<<SIZE/THREADS_PER_BLOCK, THREADS_PER_BLOCK>>>(d_minC, d_maxC, d_c, d_MBSet);

  //Copy from Device to Host
  hipMemcpy(MBSet, d_MBSet, SIZE*sizeof(int), hipMemcpyDeviceToHost);
  hipMemcpy(c, d_c, SIZE*sizeof(Complex), hipMemcpyDeviceToHost);  

  InitializeColors();
  glutPostRedisplay();   // repaint the window
}


void mouse_motion(int x, int y)
{
  if( (x<0) || (y<0) || (x>glutGet(GLUT_WINDOW_WIDTH)) || (y>glutGet(GLUT_WINDOW_HEIGHT)) )
  {
    return;
  }

  if(true == left_click)
  {
    draw_box = true;

    x2draw = x;
    y2draw = y;

    int length;

    if( ((x1draw-x2draw)*(x1draw-x2draw)) < ((y1draw-y2draw)*(y1draw-y2draw)) )
    {
      if(x1draw > x2draw)
      {
        length = x1draw - x2draw;
      }
      else
      {
        length = x2draw - x1draw;
      }

      if(y2draw > y1draw)
      {
        y2draw = y1draw + length;
      }
      else
      {
        y2draw = y1draw - length;
      }

    }
    else
    {
      if(y1draw > y2draw)
      {
        length = y1draw - y2draw;
      }
      else
      {
        length = y2draw - y1draw;
      }

      if(x2draw > x1draw)
      {
        x2draw = x1draw + length;
      }
      else
      {
        x2draw = x1draw - length;
      }
    }
  }

  glutPostRedisplay();   // repaint the window
  //cout << "X1: " << x1draw << " Y1: " << y1draw << " X2: " << x2draw << " Y2: " << y2draw << endl;
}


void mouse_click(int button, int state, int x, int y )
{

  if( (x<0) || (y<0) || (x>glutGet(GLUT_WINDOW_WIDTH)) || (y>glutGet(GLUT_WINDOW_HEIGHT)) )
  {
    //Stop drawing box if Left button is release outside window
    if( (GLUT_LEFT_BUTTON == button) && (GLUT_UP == state) )
    {
      draw_box = false;
      left_click = false;

      re_calc_draw_MBSet();
    }

    return;
  }

  if( (GLUT_LEFT_BUTTON == button) && (GLUT_DOWN == state) )
  {
    left_click = true;
    x1draw = x;
    y1draw = y;
  }
  else if( (GLUT_LEFT_BUTTON == button) && (GLUT_UP == state) )
  {
    draw_box = false;
    left_click = false;

    re_calc_draw_MBSet();
  }
}

//opengl functions end



int main(int argc, char** argv)
{
  // Initialize OPENGL here
  glutInit(&argc, argv);
  glutInitDisplayMode(GLUT_DOUBLE | GLUT_RGB);
  glutInitWindowSize(WINDOW_DIM_X, WINDOW_DIM_Y);

  GLsizei windowX = (glutGet(GLUT_SCREEN_WIDTH)-WINDOW_DIM_X)/2;
  GLsizei windowY = (glutGet(GLUT_SCREEN_HEIGHT)-WINDOW_DIM_Y)/2;
  glutInitWindowPosition(windowX, windowY);

  windowID = glutCreateWindow("Mandelbrot Set");
  glDisable(GL_DEPTH_TEST);
  glShadeModel(GL_FLAT);

  // Set up necessary host and device buffers
  c = (Complex*) malloc(SIZE*sizeof(Complex));
  MBSet = (int*) malloc(SIZE*sizeof(int));

  hipMalloc((void**)&d_c, SIZE*sizeof(Complex));
  hipMalloc((void**)&d_MBSet, SIZE*sizeof(int));
  hipMalloc((void**)&d_minC, sizeof(Complex));
  hipMalloc((void**)&d_maxC, sizeof(Complex));  

  //Copy from Host to device
  hipMemcpy(d_minC, &minC, sizeof(Complex), hipMemcpyHostToDevice);
  hipMemcpy(d_maxC, &maxC, sizeof(Complex), hipMemcpyHostToDevice);
  hipMemcpy(d_MBSet, MBSet, SIZE*sizeof(int), hipMemcpyHostToDevice);

  // Calculate the interation counts
  //Launch MBS calculation on GPU
  calcMBS_limit<<<SIZE/THREADS_PER_BLOCK, THREADS_PER_BLOCK>>>(d_minC, d_maxC, d_c, d_MBSet);

  //Copy from Device to Host
  hipMemcpy(MBSet, d_MBSet, SIZE*sizeof(int), hipMemcpyDeviceToHost);
  hipMemcpy(c, d_c, SIZE*sizeof(Complex), hipMemcpyDeviceToHost);


  glutDisplayFunc(display);
  glutIdleFunc(display);
  glutKeyboardFunc (keyboard);
  glutMouseFunc(mouse_click);
  glutMotionFunc(mouse_motion);
  glutReshapeFunc(reshape);

  // Grad students, pick the colors for the 0 .. 1999 iteration count pixels

  InitializeColors();
  glutMainLoop(); // THis will callback the display, keyboard and mouse

  free(c);
  free(MBSet);
  delete(colors);

  vector<Complex*>::iterator it_c = vec_c.begin();
  while(it_c != vec_c.end())
  {
    free(*it_c);
    it_c++;
  }

  vector<int*>::iterator it_MBSet = vec_MBSet.begin();
  while(it_MBSet != vec_MBSet.end())
  {
    free(*it_MBSet);
    it_MBSet++;
  }

  hipFree(d_c);
  hipFree(d_MBSet);
  hipFree(d_minC);
  hipFree(d_maxC);

  return 0;
}

